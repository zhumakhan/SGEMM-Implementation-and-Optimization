#include "hip/hip_runtime.h"
#include "utils.cpp"

// Unlike Tiling, matrix B isn't need to be loaded into shared memory.
// We calculate the outer product of Asub and Bsub, where the size of
// Bsub is define by TILE_SIZE and VECTOR_SIZE. 
// Specifically: 
//   Asub: TILE_SIZE * TILE_SIZE 
//   Bsub: TILE_SIZE * (TILE_SIZE*VECTOR_SIZE)
const int TILE_SIZE = 16;
const int VECTOR_SIZE = 4;

template <typename T>
__global__ void matmul_CompOpt(T *A, T *B, T *C, int M, int K, int N) {
	/* Computation method optimization.
	 * Peform outer product instead of inner product to reduce  
	 * instructions from shared memory from two to one.
	 */
	int bx = blockIdx.x, by = blockIdx.y;
	int tx = threadIdx.x, ty = threadIdx.y;

	// Explicitly allocate As as column-major array 
	// to store TILE*TILE submatrix of A.
	__shared__ T As[TILE_SIZE * TILE_SIZE];

	// Allocate register files for sub-result of C at each thread.
	T cv[TILE_SIZE] = {0};

	// Basic iterations is similar with Tiling. But notice that 
	// the total number of threads is less than that of Tiling.
	int aBegin = K * TILE_SIZE * by;
	int aEnd = aBegin + K - 1;
	int aStep = TILE_SIZE;

	int bBegin = TILE_SIZE * VECTOR_SIZE * bx;
	int bStep = TILE_SIZE * N;

	for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
		// Load Asub with size of TILE*TILE in colomn-major style.
		// Each thread needs to load TILE_SIZE / VECTOR_SIZE values of A.
		int t = VECTOR_SIZE;
		for (int i = 0; i < TILE_SIZE / VECTOR_SIZE; ++i) {
			As[ (i*t+ty) + TILE_SIZE * tx] = A[a + K*(i*t+ty) + tx];
		}
		__syncthreads();

		T *ap = As;	// Point to the first address of As, increase later.
		// TODO: global memory ? register ? not clear :(
		T *bp = &B[b + TILE_SIZE * ty + tx];	

		for (int i = 0; i < TILE_SIZE; ++i) {
			T bv = *bp;	
		// Each thread calculate a vector of C with size of TILE_SIZE.
			for (int j = 0; j < TILE_SIZE; ++j) {
				cv[j] += ap[j] * bv;
			}
			ap += TILE_SIZE;
			bp += N;
		}
		__syncthreads();
	}
	
	// Store each value of Csub back to C in global memory.
	int c = N * TILE_SIZE * by + TILE_SIZE * VECTOR_SIZE * bx;
	c += TILE_SIZE * ty + tx;
	for (int i = 0; i < TILE_SIZE; ++i) {
		C[c] = cv[i];
		c += N;
	}
}

int main(int argc, char *argv[]) {
	int M = std::atoi(argv[1]);
	int K = std::atoi(argv[2]);
	int N = std::atoi(argv[3]);

	printf("M=%d K=%d N=%d\n",M,K,N);

	dim3 threads(TILE_SIZE, VECTOR_SIZE);
	dim3 grid(N / (TILE_SIZE * VECTOR_SIZE), M / TILE_SIZE);

	float *a = utils::random_matrix_gpu<float>(M, K, utils::ROW_MAJOR);
	float *b = utils::random_matrix_gpu<float>(K, N, utils::ROW_MAJOR);
	float *c = (float*)malloc(sizeof(float)*M*N);

	float ms;
	float *dev_a, *dev_b, *dev_c;

	hipMalloc((void**)&dev_a, M*K*sizeof(float));
	hipMalloc((void**)&dev_b, K*N*sizeof(float));
	hipMalloc((void**)&dev_c, M*N*sizeof(float));

	hipMemcpy(dev_a, a, M*K*sizeof(float), hipMemcpyHostToDevice);	
	hipMemcpy(dev_b, b, K*N*sizeof(float), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
	
	matmul_CompOpt<float><<<grid, threads>>>(dev_a, dev_b, dev_c, M, K, N);

	hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipError_t cuda_error = hipGetLastError();
    if(cuda_error != hipSuccess)
    {
      printf("CUDA error: %s\n", hipGetErrorString(cuda_error));
    }

	hipMemcpy(c, dev_c, M*N*sizeof(float), hipMemcpyDeviceToHost);
#ifdef CHECK
	std::cout << (utils::check_mul<float>(a, b, c, M, K, N, utils::ROW_MAJOR,utils::ROW_MAJOR,utils::ROW_MAJOR) ? "Correct!!" : "Wrong Answer!") << std::endl;
#endif
#ifdef DEBUG
    std::cout << "Matrix A:" << std::endl;
    utils::print_mat_gpu(a, M, K, utils::ROW_MAJOR);
    std::cout << "Matrix B:" << std::endl;
    utils::print_mat_gpu(b, K, N, utils::ROW_MAJOR);
    std::cout << "Matrix C:" << std::endl;
    utils::print_mat_gpu(c, M, N, utils::ROW_MAJOR);
#endif

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    free(a);
    free(b);
    free(c);

    printf("%f\n",ms);
    
	return 0;
}


