#include "hip/hip_runtime.h"
/*
    by zhumakhan.nazir@nu.edu.kz
*/

#include "utils.cpp"
#include <stdio.h>
#include <algorithm>

#define TILE_SIZE 16
#define VECTOR_SIZE 4

/*
    A is a row major matrix ( M x K )
    B is a row major matrix ( K x N )

    Asub: TILE_SIZE x TILE_SIZE
    Bsub: TILE_SIZE x ( TILE_SIZE * VECTOR_SIZE )

    dim3 threads ( TILE_SIZE, VECTOR_SIZE )
    dim3 blocks  ( N / ( TILE_SIZE * VECTOR_SIZE ), M / TILE_SIZE )
*/

__global__ void mmPrefetching(float *A, float *B, float *C, const int M, const int K, const int N){
    
    const int bx = blockIdx.x;
    const int by = blockIdx.y;

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    const int aBegin  = TILE_SIZE * K * by;
    const int aEnd    = aBegin + K;
    const int aStep   = TILE_SIZE;

    const int bBegin  = TILE_SIZE * VECTOR_SIZE * bx;
    const int bStep   = TILE_SIZE * N;

    __shared__ float As1[ TILE_SIZE * TILE_SIZE ];
    __shared__ float As2[ TILE_SIZE * TILE_SIZE ];

    float Cv[ TILE_SIZE ] = { 0 };

    int i, j;
    
    float *ptr1, *ptr2;
    float bValue;

    // to avoid repeated computations 
    const int t1 = tx * TILE_SIZE + ty;
    const int t2 = ty * K + tx;
    const int t3 = ty * TILE_SIZE + tx;
    const int t4 = TILE_SIZE / VECTOR_SIZE;
    int t10      = 0;

    float *pre1 = As1;
    float *pre2 = As2;

// prefecth to first shared memory block;
    ptr1 = &pre1[ t1 ];
    ptr2 = &A[ aBegin + t2 ];

    for(i = 0; i < t4; ++i){
        ptr1[ t10 ] = ptr2[ t10 * K ];
        t10         += VECTOR_SIZE;
    }
    __syncthreads();



    for(int a = aBegin, b = bBegin; a < aEnd; a += aStep, b += bStep){
        
        ptr1    = &pre2[ t1 ];
        ptr2    = &A[ a + aStep + t2 ];
        t10     = 0;
        for(i = 0; i < t4; ++i){
            // load elements to As in column major way from matrix A
            ptr1[ t10 ] = ptr2[ t10 * K ];
            t10         += VECTOR_SIZE;
        }

        ptr1 = pre1;
        ptr2 = &B[ b + t3 ];

        for(i = 0; i < TILE_SIZE; ++i){
            bValue = *ptr2;

            for(j = 0; j < TILE_SIZE; ++j){
                Cv[ j ] += ptr1[ j ] * bValue;
            }

            ptr1 += TILE_SIZE;
            ptr2 += N;
        }

        __syncthreads();

//swap pointers to shared spaces
        ptr1 = pre1;
        pre1 = pre2;
        pre2 = ptr1;

    }

    j = N * TILE_SIZE * by + TILE_SIZE * VECTOR_SIZE * bx;
    // c += TILE_SIZE * ty + tx;
    j += t3;

    for(i = 0; i < TILE_SIZE; ++i){
        C[ j ] = Cv[ i ];
        j += N;
    }
}


int main(int argc, char *argv[]){
    int M = std::atoi(argv[1]);
    int K = std::atoi(argv[2]);
    int N = std::atoi(argv[3]);

    printf("M=%d K=%d N=%d\n",M,K,N);

    float *A = utils::random_matrix_gpu<float>(M, K, utils::ROW_MAJOR,-50,50);
    float *B = utils::random_matrix_gpu<float>(K, N, utils::ROW_MAJOR,-50,50);
    float *C = (float*)malloc(sizeof(float)*M*N);
    
    float ms;
    float *dA, *dB, *dC;

    hipMalloc((void**)&dA,sizeof(float)*M*K);
    hipMalloc((void**)&dB,sizeof(float)*K*N);
    hipMalloc((void**)&dC,sizeof(float)*N*M);

    hipMemcpy(dA,A,sizeof(float)*M*K, hipMemcpyHostToDevice);
    hipMemcpy(dB,B,sizeof(float)*K*N, hipMemcpyHostToDevice);

    dim3 threads( TILE_SIZE, VECTOR_SIZE );
    dim3 blocks(N / (TILE_SIZE * VECTOR_SIZE), M / TILE_SIZE);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    mmPrefetching<<<blocks,threads>>>(dA,dB,dC,M,K,N);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    hipError_t cuda_error = hipGetLastError();
    if(cuda_error != hipSuccess)
    {
      printf("CUDA error: %s\n", hipGetErrorString(cuda_error));
      exit(-1);
    }

    hipMemcpy(C,dC,sizeof(float)*N*M, hipMemcpyDeviceToHost);

#ifdef CHECK
    std::cout << (utils::check_mul<float>(A, B, C, M, K, N, utils::ROW_MAJOR, utils::ROW_MAJOR, utils::ROW_MAJOR) 
            ? "Correct!!" : "Wrong Answer!") << std::endl;
#endif
#ifdef DEBUG
    std::cout << "Matrix A:" << std::endl;
    utils::print_mat_gpu(a, M, K, utils::ROW_MAJOR);
    std::cout << "Matrix B:" << std::endl;
    utils::print_mat_gpu(b, K, N, utils::ROW_MAJOR);
    std::cout << "Matrix C:" << std::endl;
    utils::print_mat_gpu(c, M, N, utils::ROW_MAJOR);
#endif
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    free(A);
    free(B);
    free(C);

    printf("%f\n",ms);
    return 0;
}


