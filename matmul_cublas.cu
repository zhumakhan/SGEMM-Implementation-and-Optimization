#include "utils.cpp"
#include "hipblas.h"
#include <hip/hip_runtime.h>

int main(int argc, char *argv[]) {
    int M = std::atoi(argv[1]), K = std::atoi(argv[2]), N = std::atoi(argv[3]);
    float *a = utils::random_matrix_gpu<float>(M, K, utils::COLUMN_MAJOR);
    float *b = utils::random_matrix_gpu<float>(K, N, utils::COLUMN_MAJOR);
    float *c = (float*)malloc(M*N*sizeof(float));

    float *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, M*K*sizeof(float));
    hipMalloc((void**)&dev_b, K*N*sizeof(float));
    hipMalloc((void**)&dev_c, M*N*sizeof(float));

    hipMemcpy(dev_a, a, M*K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, K*N*sizeof(float), hipMemcpyHostToDevice);
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float al=1.0f, bet=0;
    // hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, 
	//	    &al, dev_a, M, dev_b, K, &bet, dev_c, M);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, 
		    &al, dev_a, M, dev_b, K, &bet, dev_c, M);

    hipMemcpy(c, dev_c, M*N*sizeof(float), hipMemcpyDeviceToHost);
#ifdef CHECK
    std::cout << (utils::check_mul<float>(a, b, c, M, K, N, utils::COLUMN_MAJOR) 
		    ? "Correct!!" : "Wrong Answer!") << std::endl;
#endif
#ifdef DEBUG
    std::cout << "Matrix A:" << std::endl;
    utils::print_mat_gpu(a, M, K, utils::COLUMN_MAJOR);
    std::cout << "\nMatrix B:" << std::endl;
    utils::print_mat_gpu(b, K, N, utils::COLUMN_MAJOR);
    std::cout << "\nMatrix C:" << std::endl;
    utils::print_mat_gpu(c, M, N, utils::COLUMN_MAJOR);
#endif

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    free(a);
    free(b);
    free(c);
    return 0;
}
    
