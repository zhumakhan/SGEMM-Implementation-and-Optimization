#include "utils.cpp"
#include <hipblas.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

int main(int argc, char *argv[]) {
    int M = std::atoi(argv[1]), K = std::atoi(argv[2]), N = std::atoi(argv[3]);
    printf("M=%d K=%d N=%d\n",M,K,N);

    float *a = utils::random_matrix_gpu<float>(M, K, utils::COLUMN_MAJOR,-50,50);
    float *b = utils::random_matrix_gpu<float>(K, N, utils::COLUMN_MAJOR,-50,50);
    float *c = (float*)malloc(M*N*sizeof(float));

    float *dev_a, *dev_b, *dev_c;
    float ms;

    hipMalloc((void**)&dev_a, M*K*sizeof(float));
    hipMalloc((void**)&dev_b, K*N*sizeof(float));
    hipMalloc((void**)&dev_c, M*N*sizeof(float));

    hipMemcpy(dev_a, a, M*K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, K*N*sizeof(float), hipMemcpyHostToDevice);
    
    hipblasStatus_t status;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float alpha=1.0f, beta=0;
    // hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, 
	//	    &al, dev_a, M, dev_b, K, &bet, dev_c, M);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, 
		    &alpha, dev_a, M, dev_b, K, &beta, dev_c, M);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    hipError_t cuda_error = hipGetLastError();
    if(cuda_error != hipSuccess)
    {
      printf("CUDA error: %s\n", hipGetErrorString(cuda_error));
      exit(-1);
    }

    switch(status){
        case (HIPBLAS_STATUS_SUCCESS):{
            break;
        }
        case (HIPBLAS_STATUS_NOT_INITIALIZED):{
            printf("the library was not initialized\n");
            break;
        }
        case (HIPBLAS_STATUS_INVALID_VALUE):{
            printf("the parameters m,n,k<0\n");
            break;
        }
        case (HIPBLAS_STATUS_ARCH_MISMATCH):{
            printf("Architecture problem. in the case of hipblasHgemm the device does not support math in half precision.\n");
            break;
        }
        case (HIPBLAS_STATUS_EXECUTION_FAILED):{
            printf("the function failed to launch on the GPU\n");
            break;
        }
        default:{
            printf("Unknown error occured in hipblasSgemm\n");
            break;
        }
    }

    hipMemcpy(c, dev_c, M*N*sizeof(float), hipMemcpyDeviceToHost);
#ifdef CHECK
    std::cout << (utils::check_mul<float>(a, b, c, M, K, N, utils::COLUMN_MAJOR, utils::COLUMN_MAJOR, utils::COLUMN_MAJOR) 
		    ? "Correct!!" : "Wrong Answer!") << std::endl;
#endif
#ifdef DEBUG
    std::cout << "Matrix A:" << std::endl;
    utils::print_mat_gpu(a, M, K, utils::COLUMN_MAJOR);
    std::cout << "\nMatrix B:" << std::endl;
    utils::print_mat_gpu(b, K, N, utils::COLUMN_MAJOR);
    std::cout << "\nMatrix C:" << std::endl;
    utils::print_mat_gpu(c, M, N, utils::COLUMN_MAJOR);
#endif

    hipblasDestroy(handle);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    free(a);
    free(b);
    free(c);
    
    printf("%f\n",ms);
    return 0;
}
    
