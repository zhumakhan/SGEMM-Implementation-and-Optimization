/*
    by zhumakhan.nazir@nu.edu.kz
*/

#include "utils.cpp"
#include "tiling_with_shared_memory.cu"
#include "computation_optimization.cu"
#include "loop_unrolling.cu"
#include "prefetching.cu"

#include <stdio.h>

#define TILE_SIZE 16
#define VECTOR_SIZE 4

/*
    A is a row major matrix ( M x K )
    B is a row major matrix ( K x N )

*/

typedef void (*FunctionPointer_t)(float *, float *, float *, const int, const int, const int);

class Kernel_t{
public:
    std::string name;
    FunctionPointer_t function;
    dim3 threads;
    dim3 blocks;

    Kernel_t(std::string name, FunctionPointer_t function, dim3 threads, dim3 blocks):name(name),function(function), threads(threads), blocks(blocks){
    }
};


int main(int argc, char *argv[]){
    int M = std::atoi(argv[1]);
    int K = std::atoi(argv[2]);
    int N = std::atoi(argv[3]);

    printf("M=%d K=%d N=%d\n",M,K,N);

    float *A = utils::random_matrix_gpu<float>(M, K, utils::ROW_MAJOR,-50,50);
    float *B = utils::random_matrix_gpu<float>(K, N, utils::ROW_MAJOR,-50,50);
    float *C = (float*)malloc(sizeof(float)*M*N);
    
    float ms;
    float *dA, *dB, *dC;

    hipEvent_t start, stop;

    hipMalloc((void**)&dA,sizeof(float)*M*K);
    hipMalloc((void**)&dB,sizeof(float)*K*N);
    hipMalloc((void**)&dC,sizeof(float)*N*M);

    hipMemcpy(dA,A,sizeof(float)*M*K, hipMemcpyHostToDevice);
    hipMemcpy(dB,B,sizeof(float)*K*N, hipMemcpyHostToDevice);

    dim3 threads1( TILE_SIZE, TILE_SIZE );
    dim3 blocks1(N / TILE_SIZE, M / TILE_SIZE);

    dim3 threads2( TILE_SIZE, VECTOR_SIZE );
    dim3 blocks2(N / (TILE_SIZE * VECTOR_SIZE), M / TILE_SIZE);
    
    Kernel_t kernels [ 4 ] = {
        {   "tiling", &mmSharedRR, threads1, blocks1      },
        {   "comp_opt", &mmCompOpt_v1, threads2, blocks2    },
        {   "unrolling", &mmLoopUnrolling, threads2, blocks2 },
        {   "prefetching", &mmPrefetching, threads2, blocks2   }
    };

    for(int i = 0; i < 4; i++){

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        
        (*kernels[i].function) <<< kernels[i].blocks, kernels[i].threads >>> ( dA, dB, dC, M, K, N );
        
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        
        hipError_t cuda_error = hipGetLastError();
        if(cuda_error != hipSuccess)
        {
          printf("CUDA error: %s\n", hipGetErrorString(cuda_error));
          exit(-1);
        }

        hipMemcpy(C,dC,sizeof(float)*N*M, hipMemcpyDeviceToHost);

        #ifdef CHECK
        std::cout << (utils::check_mul<float>(A, B, C, M, K, N, utils::ROW_MAJOR, utils::ROW_MAJOR, utils::ROW_MAJOR) 
                ? "Correct!!" : "Wrong Answer!") << std::endl;
        #endif

        std::cout << kernels[i].name << " " << ms << std::endl;
    }


#ifdef DEBUG
    std::cout << "Matrix A:" << std::endl;
    utils::print_mat_gpu(a, M, K, utils::ROW_MAJOR);
    std::cout << "Matrix B:" << std::endl;
    utils::print_mat_gpu(b, K, N, utils::ROW_MAJOR);
    std::cout << "Matrix C:" << std::endl;
    utils::print_mat_gpu(c, M, N, utils::ROW_MAJOR);
#endif

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    free(A);
    free(B);
    free(C);

    return 0;
}
