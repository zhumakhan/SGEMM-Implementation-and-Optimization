/*
    by zhumakhan.nazir@nu.edu.kz
*/

#include "utils.cpp"
#include "test_cublas.cu"
#include "tiling_with_shared_memory.cu"
#include "computation_optimization.cu"
#include "loop_unrolling.cu"
#include "prefetching.cu"

#include <stdio.h>
#include <unistd.h>

#define TILE_SIZE 16
#define VECTOR_SIZE 4

/*
    A is a row major matrix ( M x K )
    B is a row major matrix ( K x N )

*/

typedef void (*FunctionPointer_t)(float *, float *, float *, const int, const int, const int);

class Kernel_t{
public:
    std::string name;
    FunctionPointer_t function;
    dim3 threads;
    dim3 blocks;

    Kernel_t(std::string name, FunctionPointer_t function, dim3 threads, dim3 blocks):name(name),function(function), threads(threads), blocks(blocks){
    }
};


int main(int argc, char *argv[]){
    const int M = std::atoi(argv[1]);
    const int K = std::atoi(argv[2]);
    const int N = std::atoi(argv[3]);

    printf("M=%d K=%d N=%d\n",M,K,N);

    
    test_cublas( M, K, N );
    

    float *A = utils::random_matrix_gpu<float>(M, K, utils::ROW_MAJOR,-50,50);
    float *B = utils::random_matrix_gpu<float>(K, N, utils::ROW_MAJOR,-50,50);
    float *C = (float*)malloc(sizeof(float)*M*N);
    
    float ms;
    float *dA, *dB, *dC;

    hipEvent_t start, stop;

    hipMalloc((void**)&dA,sizeof(float)*M*K);
    hipMalloc((void**)&dB,sizeof(float)*K*N);
    hipMalloc((void**)&dC,sizeof(float)*N*M);

    hipMemcpy(dA,A,sizeof(float)*M*K, hipMemcpyHostToDevice);
    hipMemcpy(dB,B,sizeof(float)*K*N, hipMemcpyHostToDevice);

    const dim3 threads1( TILE_SIZE, TILE_SIZE );
    const dim3 blocks1(N / TILE_SIZE, M / TILE_SIZE);

    const dim3 threads2( TILE_SIZE, VECTOR_SIZE );
    const dim3 blocks2(N / (TILE_SIZE * VECTOR_SIZE), M / TILE_SIZE);
    
    const Kernel_t kernels [ 4 ] = {
        {   "tiling", &mmSharedRR, threads1, blocks1      },
        {   "comp_opt", &mmCompOpt_v1, threads2, blocks2    },
        {   "unrolling", &mmLoopUnrolling, threads2, blocks2 },
        {   "prefetching", &mmPrefetching, threads2, blocks2   }
    };

    for(int i = 0; i < 4; i++){

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        
        sleep(5);

        (*kernels[i].function) <<< kernels[i].blocks, kernels[i].threads >>> ( dA, dB, dC, M, K, N );
        
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        
        hipError_t cuda_error = hipGetLastError();
        if(cuda_error != hipSuccess)
        {
          printf("CUDA error: %s\n", hipGetErrorString(cuda_error));
          exit(-1);
        }

        bool correct = true;
        
#ifdef CHECK
        hipMemcpy(C,dC,sizeof(float)*N*M, hipMemcpyDeviceToHost);
        correct = utils::check_mul<float>(A, B, C, M, K, N, utils::ROW_MAJOR, utils::ROW_MAJOR, utils::ROW_MAJOR);
#endif

        std::cout << kernels[i].name << " " << ms << " " << correct << std::endl;
    }


#ifdef DEBUG

#ifndef CHECK
    hipMemcpy(C,dC,sizeof(float)*N*M, hipMemcpyDeviceToHost);
#endif

    std::cout << "Matrix A:" << std::endl;
    utils::print_mat_gpu(a, M, K, utils::ROW_MAJOR);
    std::cout << "Matrix B:" << std::endl;
    utils::print_mat_gpu(b, K, N, utils::ROW_MAJOR);
    std::cout << "Matrix C:" << std::endl;
    utils::print_mat_gpu(c, M, N, utils::ROW_MAJOR);
#endif

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    free(A);
    free(B);
    free(C);

    return 0;
}
