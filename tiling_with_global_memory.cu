#include "hip/hip_runtime.h"
#include "utils.cpp"
#include <stdio.h>
// #define N 1024
// #define P 512
// #define M 256
/*
#define N 512
#define P 256
#define M 128
*/
#define IDX(i,j,row,col) (i*col+j)
#define Tx 1
#define Ty 64 //max value for Tx * Ty is 1024

__global__ void mmGlobal(float *A, float *B, float *C, int M, int K, int N);

int main(int argc, char *argv[]){
    int M = std::atoi(argv[1]), K = std::atoi(argv[2]), N = std::atoi(argv[3]);
    printf("M=%d K=%d N=%d\n",M,K,N);

    float *A = utils::random_matrix_gpu<float>(M, K, utils::ROW_MAJOR,-50,50);
    float *B = utils::random_matrix_gpu<float>(K, N, utils::ROW_MAJOR,-50,50);
    float *C = (float*)malloc(sizeof(float)*M*N);
    
    float ms;
    float *dA, *dB, *dC;

    hipMalloc((void**)&dA,sizeof(float)*M*K);
    hipMalloc((void**)&dB,sizeof(float)*K*N);
    hipMalloc((void**)&dC,sizeof(float)*N*M);

    hipMemcpy(dA,A,sizeof(float)*M*K, hipMemcpyHostToDevice);
    hipMemcpy(dB,B,sizeof(float)*K*N, hipMemcpyHostToDevice);

    dim3 threads(Tx,Ty);
    dim3 blocks( (M+threads.x-1)/threads.x, (N+threads.y-1)/threads.y);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    mmGlobal<<<blocks,threads>>>(dA,dB,dC,M,K,N);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    hipError_t cuda_error = hipGetLastError();
    if(cuda_error != hipSuccess)
    {
      printf("CUDA error: %s\n", hipGetErrorString(cuda_error));
      exit(-1);
    }

    hipMemcpy(C,dC,sizeof(float)*N*M, hipMemcpyDeviceToHost);

#ifdef CHECK
    std::cout << (utils::check_mul<float>(A, B, C, M, K, N, utils::ROW_MAJOR) 
            ? "Correct!!" : "Wrong Answer!") << std::endl;
#endif

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    free(A);
    free(B);
    free(C);

    printf("%f\n",ms);
    return 0;
}
__global__ void mmGlobal(float *A, float *B, float *C, int M, int K, int N){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    float temp = 0;
    if(i < M and j < N){
        for(int k = 0; k < K; ++k){
          temp += A[ IDX(i,k,M,K) ] * B[ IDX(k,j,K,N) ];
      }
      C[ IDX(i,j,M,N) ]=temp;
    }
}