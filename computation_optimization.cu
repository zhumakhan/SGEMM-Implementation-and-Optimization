#include "hip/hip_runtime.h"
/*
    by zhumakhan.nazir@nu.edu.kz
*/

#include "utils.cpp"
#include <stdio.h>

#define TILE_SIZE 16
#define VECTOR_SIZE 4

/*
    A is a row major matrix ( M x K )
    B is a row major matrix ( K x N )

    Asub: TILE_SIZE x TILE_SIZE
    Bsub: TILE_SIZE x ( TILE_SIZE * VECTOR_SIZE )

    dim3 threads ( TILE_SIZE, VECTOR_SIZE )
    dim3 blocks  ( N / ( TILE_SIZE * VECTOR_SIZE ), M / TILE_SIZE )
*/

__global__ void mmCompOpt(float *A, float *B, float *C, int M, int K, int N){
    
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    __shared__ float As[ TILE_SIZE * TILE_SIZE ];

    float Cv[ TILE_SIZE ] = { 0 };

    int aBegin  = K * TILE_SIZE * by;
    int aEnd    = aBegin + K;
    int aStep   = TILE_SIZE;

    int bBegin  = TILE_SIZE * VECTOR_SIZE * bx;
    int bStep   = TILE_SIZE * N;

    int i, j;
    
    float *aPtr, *bPtr;
    float bValue;

    // to avoid repeated computations 
    // int t1 = tx * TILE_SIZE + ty;
    // int t2 = ty * K + tx;
    // int t3 = ty * TILE_SIZE + tx;
    // int t4;

    for(int a = aBegin, b = bBegin; a < aEnd; a += aStep, b += bStep){

        for(i = 0; i < TILE_SIZE / VECTOR_SIZE; ++i){
            // load elements to As in column major way from matrix A
            // t4 = i * VECTOR_SIZE;
            As[ tx * TILE_SIZE + ty + i * VECTOR_SIZE ] = A[ a + K * (i * VECTOR_SIZE + ty) + tx ];
            // As[ t1 + t4 ] = A[ a + t4 * K + t2 ];
        }
        
        __syncthreads();

        aPtr = As;
        bPtr = &B[ b + TILE_SIZE * ty + tx ];
        // bPtr = &B[b + t3 ];

        for(i = 0; i < TILE_SIZE; ++i){
            bValue = *bPtr;

            for(j = 0; j < TILE_SIZE; ++j){
                Cv[ j ] += aPtr[ j ] * bValue;
            }

            aPtr += TILE_SIZE;
            bPtr += N;
        }

        __syncthreads();

    }

    int c = N * TILE_SIZE * by + TILE_SIZE * VECTOR_SIZE * bx;
    c += TILE_SIZE * ty + tx;
    // c += t3;

    for(i = 0; i < TILE_SIZE; ++i){
        C[ c ] = Cv[ i ];
        c += N;
    }
}

__global__ void mmCompOpt_v1(float *A, float *B, float *C, const int M, const int K, const int N){
    
    const int bx = blockIdx.x;
    const int by = blockIdx.y;

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    __shared__ float As[ TILE_SIZE * TILE_SIZE ];

    float Cv[ TILE_SIZE ] = { 0 };

    const int aBegin  = K * TILE_SIZE * by;
    const int aEnd    = aBegin + K;
    const int aStep   = TILE_SIZE;

    const int bBegin  = TILE_SIZE * VECTOR_SIZE * bx;
    const int bStep   = TILE_SIZE * N;

    int i, j;
    
    float *aPtr, *bPtr;
    float bValue;

    // to avoid repeated computations 
    const int t1 = tx * TILE_SIZE + ty;
    const int t2 = ty * K + tx;
    const int t3 = ty * TILE_SIZE + tx;
    int t4;
    const int t5 = TILE_SIZE / VECTOR_SIZE;

    for(int a = aBegin, b = bBegin; a < aEnd; a += aStep, b += bStep){

        for(i = 0; i < t5; ++i){
            // load elements to As in column major way from matrix A
            t4 = i * VECTOR_SIZE;
            // As[ tx * TILE_SIZE + ty + i * VECTOR_SIZE ] = A[ a + K * (i * VECTOR_SIZE + ty) + tx ];
            As[ t1 + t4 ] = A[ a + t4 * K + t2 ];
        }
        
        __syncthreads();

        aPtr = As;
        // bPtr = &B[ b + TILE_SIZE * ty + tx ];
        bPtr = &B[b + t3 ];

        for(i = 0; i < TILE_SIZE; ++i){
            bValue = *bPtr;

            for(j = 0; j < TILE_SIZE; ++j){
                Cv[ j ] += aPtr[ j ] * bValue;
            }

            aPtr += TILE_SIZE;
            bPtr += N;
        }

        __syncthreads();

    }

    int c = N * TILE_SIZE * by + TILE_SIZE * VECTOR_SIZE * bx;
    // c += TILE_SIZE * ty + tx;
    c += t3;

    for(i = 0; i < TILE_SIZE; ++i){
        C[ c ] = Cv[ i ];
        c += N;
    }
}

int main(int argc, char *argv[]){
    int M = std::atoi(argv[1]);
    int K = std::atoi(argv[2]);
    int N = std::atoi(argv[3]);

    printf("M=%d K=%d N=%d\n",M,K,N);

    float *A = utils::random_matrix_gpu<float>(M, K, utils::ROW_MAJOR,-50,50);
    float *B = utils::random_matrix_gpu<float>(K, N, utils::ROW_MAJOR,-50,50);
    float *C = (float*)malloc(sizeof(float)*M*N);
    
    float ms;
    float *dA, *dB, *dC;

    hipMalloc((void**)&dA,sizeof(float)*M*K);
    hipMalloc((void**)&dB,sizeof(float)*K*N);
    hipMalloc((void**)&dC,sizeof(float)*N*M);

    hipMemcpy(dA,A,sizeof(float)*M*K, hipMemcpyHostToDevice);
    hipMemcpy(dB,B,sizeof(float)*K*N, hipMemcpyHostToDevice);

    dim3 threads( TILE_SIZE, VECTOR_SIZE );
    dim3 blocks(N / (TILE_SIZE * VECTOR_SIZE), M / TILE_SIZE);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    mmCompOpt_v1<<<blocks,threads>>>(dA,dB,dC,M,K,N);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    hipError_t cuda_error = hipGetLastError();
    if(cuda_error != hipSuccess)
    {
      printf("CUDA error: %s\n", hipGetErrorString(cuda_error));
      exit(-1);
    }

    hipMemcpy(C,dC,sizeof(float)*N*M, hipMemcpyDeviceToHost);

#ifdef CHECK
    std::cout << (utils::check_mul<float>(A, B, C, M, K, N, utils::ROW_MAJOR, utils::ROW_MAJOR, utils::ROW_MAJOR) 
            ? "Correct!!" : "Wrong Answer!") << std::endl;
#endif
#ifdef DEBUG
    std::cout << "Matrix A:" << std::endl;
    utils::print_mat_gpu(a, M, K, utils::ROW_MAJOR);
    std::cout << "Matrix B:" << std::endl;
    utils::print_mat_gpu(b, K, N, utils::ROW_MAJOR);
    std::cout << "Matrix C:" << std::endl;
    utils::print_mat_gpu(c, M, N, utils::ROW_MAJOR);
#endif
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    free(A);
    free(B);
    free(C);

    printf("%f\n",ms);
    return 0;
}




