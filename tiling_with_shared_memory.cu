#include "hip/hip_runtime.h"
#include "utils.cpp"
#include <stdio.h>


#define BS 16

__global__ void mmShared(float *A, float *B, float *C, int M, int K, int N);

int main(int argc, char *argv[]){
    int M = std::atoi(argv[1]), K = std::atoi(argv[2]), N = std::atoi(argv[3]);
    printf("M=%d K=%d N=%d\n",M,K,N);

    float *A = utils::random_matrix_gpu<float>(M, K, utils::ROW_MAJOR,-50,50);
    float *B = utils::random_matrix_gpu<float>(K, N, utils::ROW_MAJOR,-50,50);
    float *C = (float*)malloc(sizeof(float)*M*N);
    
    float ms;
    float *dA, *dB, *dC;

    hipMalloc((void**)&dA,sizeof(float)*M*K);
    hipMalloc((void**)&dB,sizeof(float)*K*N);
    hipMalloc((void**)&dC,sizeof(float)*M*N);

    hipMemcpy(dA,A,sizeof(float)*M*K, hipMemcpyHostToDevice);
    hipMemcpy(dB,B,sizeof(float)*K*N, hipMemcpyHostToDevice);

    dim3 threads(BS,BS);
    dim3 blocks( (M+threads.x-1)/threads.x, (N+threads.y-1)/threads.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    mmShared<<<blocks,threads>>>(dA,dB,dC,M,K,N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipError_t cuda_error = hipGetLastError();
    if(cuda_error != hipSuccess)
    {
      printf("CUDA error: %s\n", hipGetErrorString(cuda_error));
    }

    hipMemcpy(C, dC, sizeof(float)*M*N, hipMemcpyDeviceToHost);

#ifdef CHECK
    std::cout << (utils::check_mul<float>(A, B, C, M, K, N, utils::ROW_MAJOR, utils::ROW_MAJOR, utils::ROW_MAJOR) 
            ? "Correct!!" : "Wrong Answer!") << std::endl;
#endif
#ifdef DEBUG
    std::cout << "Matrix A:" << std::endl;
    utils::print_mat_gpu(A, M, K, utils::ROW_MAJOR);
    std::cout << "Matrix B:" << std::endl;
    utils::print_mat_gpu(B, K, N, utils::ROW_MAJOR);
    std::cout << "Matrix C:" << std::endl;
    utils::print_mat_gpu(C, M, N, utils::ROW_MAJOR);
#endif

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    free(A);
    free(B);
    free(C);

    printf("%f\n",ms);
    return 0;
}

__global__ void mmShared(float *A, float *B, float *C, int M, int K, int N){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if( i >= M or j >= N){
        return;
    }

    int ii = threadIdx.x;
    int jj = threadIdx.y;

  
    __shared__ float sA[BS][BS], sB[BS][BS];

    float temp = 0;
    int k,m;

    for(k = 0; k < K; k += BS){
        sA[ii][jj] = A[ IDXR(i,k+jj, M, K) ];
        sB[ii][jj] = B[ IDXR(k+ii,j, K, N) ];
        
        __syncthreads();

         for(m = 0; m < BS and k+m < K; m += 1){
            temp += sA[ii][m] * sB[m][jj];
        }
        __syncthreads();
    }
    C[ IDXR(i,j,M,N) ] = temp;
}


