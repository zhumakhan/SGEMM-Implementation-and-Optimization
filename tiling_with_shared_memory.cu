#include "hip/hip_runtime.h"
/*
    by zhumakhan.nazir@nu.edu.kz
*/

#include "utils.cpp"
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mmSharedRR(float *, float *, float *, int, int, int);
__global__ void mmSharedRC(float *, float *, float *, int, int, int);
__global__ void mmSharedCR(float *, float *, float *, int, int, int);
__global__ void mmSharedCC(float *, float *, float *, int, int, int);

void test(void (*kernel)(float *, float *, float *, int, int, int),
    dim3 &threads, dim3 &blocks, int M, int K, int N, 
    int a_major, int b_major, int c_major);


int main(int argc, char *argv[]){
    int M = std::atoi(argv[1]);
    int K = std::atoi(argv[2]);
    int N = std::atoi(argv[3]);

    printf("M=%d K=%d N=%d\n",M,K,N);
    

    dim3 threads(   TILE_SIZE, TILE_SIZE    );
    dim3 blocks(    ( N + threads.x - 1 ) / threads.x, ( M + threads.y - 1 ) / threads.y   );

    std::cout << "Row Row" << std::endl;
    test(&mmSharedRR, threads, blocks, M, K, N, utils::ROW_MAJOR, utils::ROW_MAJOR, utils::ROW_MAJOR);
    std::cout << "Row Column" << std::endl;
    test(&mmSharedRC, threads, blocks, M, K, N, utils::ROW_MAJOR, utils::COLUMN_MAJOR, utils::ROW_MAJOR);
    std::cout << "Column Row" << std::endl;
    test(&mmSharedCR, threads, blocks, M, K, N, utils::COLUMN_MAJOR, utils::ROW_MAJOR, utils::ROW_MAJOR);
    std::cout << "Column Column" << std::endl;
    test(&mmSharedCC, threads, blocks, M, K, N, utils::COLUMN_MAJOR, utils::COLUMN_MAJOR, utils::ROW_MAJOR);

    return 0;
}

__global__ void mmSharedRR(float *A, float *B, float *C, int M, int K, int N){
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    volatile __shared__ float As[ TILE_SIZE ][ TILE_SIZE ];
    volatile __shared__ float Bs[ TILE_SIZE ][ TILE_SIZE ];

    int aBegin  = K * TILE_SIZE * by;
    int aEnd    = aBegin + K;
    int aStep   = TILE_SIZE;

    int bBegin  = TILE_SIZE * bx;
    int bStep   = TILE_SIZE * N;

    float Csub = 0;

    for (int i = aBegin, j = bBegin; i < aEnd; i += aStep, j += bStep) {
        As[ ty ][ tx ] = A[ i + K * ty + tx ];
        Bs[ tx ][ ty ] = B[ j + N * tx + ty ];

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) {
            Csub += As[ ty ][ k ] * Bs[ k ][ tx ];
        }
        
        __syncthreads();
    }
    int cIdx = N * TILE_SIZE * by + TILE_SIZE * bx;
    C[ cIdx + N * ty + tx ] = Csub;
}

__global__ void mmSharedRC(float *A, float *B, float *C, int M, int K, int N){
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    volatile __shared__ float As[TILE_SIZE][TILE_SIZE];
    volatile __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int aBegin  = K * TILE_SIZE * by;
    int aEnd    = aBegin + K;
    int aStep   = TILE_SIZE;

    int bBegin  = K * TILE_SIZE * bx;
    int bStep   = TILE_SIZE;

    float Csub  = 0;

    for (int i = aBegin, j = bBegin; i < aEnd; i += aStep, j += bStep) {
        As[ty][tx] = A[i + K * ty + tx];
        Bs[tx][ty] = B[j + K * ty + tx];

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) {
            Csub += As[ty][k]*Bs[k][tx];
        }
        
        __syncthreads();
    }
    int cIdx = N * TILE_SIZE * by + TILE_SIZE * bx;
    C[cIdx + N * ty + tx] = Csub;
}

__global__ void mmSharedCR(float *A, float *B, float *C, int M, int K, int N){
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    volatile __shared__ float As[TILE_SIZE][TILE_SIZE];
    volatile __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int aBegin  = TILE_SIZE * by;
    int aEnd    = aBegin + K*M;
    int aStep   = TILE_SIZE * M;

    int bBegin  = TILE_SIZE * bx;
    int bStep   = TILE_SIZE * N;

    float Csub = 0;

    for (int i = aBegin, j = bBegin; i < aEnd; i += aStep, j += bStep) {
        As[ty][tx] = A[i + M * tx + ty];
        Bs[tx][ty] = B[j + N * tx + ty];
        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) {
            Csub += As[ty][k]*Bs[k][tx];
        }
        
        __syncthreads();
    }
    int cIdx = N * TILE_SIZE * by + TILE_SIZE * bx;
    C[cIdx + N * ty + tx] = Csub;
}

__global__ void mmSharedCC(float *A, float *B, float *C, int M, int K, int N){
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    volatile __shared__ float As[TILE_SIZE][TILE_SIZE];
    volatile __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int aBegin  = TILE_SIZE * by;
    int aEnd    = aBegin + K*M;
    int aStep   = TILE_SIZE * M;

    int bBegin  = K * TILE_SIZE * bx;
    int bStep   = TILE_SIZE;

    float Csub  = 0;

    for (int i = aBegin, j = bBegin; i < aEnd; i += aStep, j += bStep) {
        As[ty][tx] = A[i + M * tx + ty];
        Bs[tx][ty] = B[j + K * ty + tx];

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) {
            Csub += As[ty][k]*Bs[k][tx];
        }
        
        __syncthreads();
    }
    int cIdx = N * TILE_SIZE * by + TILE_SIZE * bx;
    C[cIdx + N * ty + tx] = Csub;
}

void test(void (*kernel)(float *, float *, float *, int, int, int),
    dim3 &threads, dim3 &blocks, int M, int K, int N, 
    int a_major, int b_major, int c_major){
    
    float *A = utils::random_matrix_gpu<float>(M, K, a_major,-1,1);
    float *B = utils::random_matrix_gpu<float>(K, N, b_major,-1,1);
    float *C = (float*)malloc(sizeof(float)*M*N);
    
    float *dA, *dB, *dC;

    hipMalloc((void**)&dA,sizeof(float)*M*K);
    hipMalloc((void**)&dB,sizeof(float)*K*N);
    hipMalloc((void**)&dC,sizeof(float)*M*N);

    hipMemcpy(dA,A,sizeof(float)*M*K, hipMemcpyHostToDevice);
    hipMemcpy(dB,B,sizeof(float)*K*N, hipMemcpyHostToDevice);

    float ms;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    (*kernel)<<<blocks,threads>>>(dA,dB,dC,M,K,N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipError_t cuda_error = hipGetLastError();
    if(cuda_error != hipSuccess)
    {
      printf("CUDA error: %s\n", hipGetErrorString(cuda_error));
    }

    hipMemcpy(C, dC, sizeof(float)*M*N, hipMemcpyDeviceToHost);

#ifdef CHECK
    std::cout << (utils::check_mul<float>(A, B, C, M, K, N, a_major, b_major, c_major) 
            ? "Correct!!" : "Wrong Answer!") << std::endl;
#endif
#ifdef DEBUG
    std::cout << "Matrix A:" << std::endl;
    utils::print_mat_gpu(A, M, K, a_major);
    std::cout << "Matrix B:" << std::endl;
    utils::print_mat_gpu(B, K, N, b_major);
    std::cout << "Matrix C:" << std::endl;
    utils::print_mat_gpu(C, M, N, c_major);
#endif

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    free(A);
    free(B);
    free(C);

    printf("%f ms\n\n", ms);

}



